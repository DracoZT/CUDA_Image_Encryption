
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef short WORD;
typedef int DWORD;
typedef int LONG;

__device__ static unsigned char s_box[256] = {
	// 0     1     2     3     4     5     6     7     8     9     a     b     c     d     e     f
	0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76, // 0
	0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, // 1
	0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, // 2
	0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, // 3
	0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, // 4
	0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf, // 5
	0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, // 6
	0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, // 7
	0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73, // 8
	0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb, // 9
	0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, // a
	0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, // b
	0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a, // c
	0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, // d
	0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, // e
	0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16};// f

__device__ static unsigned char inv_s_box[256] = {
	// 0     1     2     3     4     5     6     7     8     9     a     b     c     d     e     f
	0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb, // 0
	0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb, // 1
	0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e, // 2
	0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25, // 3
	0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92, // 4
	0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84, // 5
	0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06, // 6
	0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b, // 7
	0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73, // 8
	0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e, // 9
	0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b, // a
	0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4, // b
	0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f, // c
	0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef, // d
	0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61, // e
	0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d};// f

__device__ int Nb = 4;

__device__ void shift_rows(unsigned char *state) {
	unsigned char i, k, s, tmp;
	for (i = 1; i < 4; i++) {
		s = 0;
		while (s < i) {
			tmp = state[Nb*i+0];
			
			for (k = 1; k < Nb; k++) {
				state[Nb*i+k-1] = state[Nb*i+k];
			}

			state[Nb*i+Nb-1] = tmp;
			s++;
		}
	}
}

__device__ void inv_shift_rows(unsigned char *state) {
	unsigned char i, k, s, tmp;
	for (i = 1; i < 4; i++) {
		s = 0;
		while (s < i) {
			tmp = state[Nb*i+Nb-1];
			
			for (k = Nb-1; k > 0; k--) {
				state[Nb*i+k] = state[Nb*i+k-1];
			}

			state[Nb*i+0] = tmp;
			s++;
		}
	}
}

__device__ unsigned char gmult(unsigned char a, unsigned char b) {

	unsigned char p = 0, i = 0, hbs = 0;

	for (i = 0; i < 8; i++) {
		if (b & 1) {
			p ^= a;
		}

		hbs = a & 0x80;
		a <<= 1;
		if (hbs) a ^= 0x1b; // 0000 0001 0001 1011	
		b >>= 1;
	}

	return (unsigned char)p;
}

__device__ void coef_mult(unsigned char *a, unsigned char *b, unsigned char *d) {

	d[0] = gmult(a[0],b[0])^gmult(a[3],b[1])^gmult(a[2],b[2])^gmult(a[1],b[3]);
	d[1] = gmult(a[1],b[0])^gmult(a[0],b[1])^gmult(a[3],b[2])^gmult(a[2],b[3]);
	d[2] = gmult(a[2],b[0])^gmult(a[1],b[1])^gmult(a[0],b[2])^gmult(a[3],b[3]);
	d[3] = gmult(a[3],b[0])^gmult(a[2],b[1])^gmult(a[1],b[2])^gmult(a[0],b[3]);
}

__device__ void mix_columns(unsigned char *state) {

	unsigned char a[] = {0x02, 0x01, 0x01, 0x03}; // a(x) = {02} + {01}x + {01}x2 + {03}x3
	unsigned char i, j, col[4], res[4];

	for (j = 0; j < Nb; j++) {
		for (i = 0; i < 4; i++) {
			col[i] = state[Nb*i+j];
		}

		coef_mult(a, col, res);

		for (i = 0; i < 4; i++) {
			state[Nb*i+j] = res[i];
		}
	}
}


__device__ void inv_mix_columns(unsigned char *state) {

	unsigned char a[] = {0x0e, 0x09, 0x0d, 0x0b}; // a(x) = {0e} + {09}x + {0d}x2 + {0b}x3
	unsigned char i, j, col[4], res[4];

	for (j = 0; j < Nb; j++) {
		for (i = 0; i < 4; i++) {
			col[i] = state[Nb*i+j];
		}

		coef_mult(a, col, res);

		for (i = 0; i < 4; i++) {
			state[Nb*i+j] = res[i];
		}
	}
}

//Round Keys
__device__ unsigned char key[16] = {
	0x00, 0x01, 0x02, 0x03,
	0x04, 0x05, 0x06, 0x07,
	0x08, 0x09, 0x0a, 0x0b,
	0x0c, 0x0d, 0x0e, 0x0f};

__device__ void key_xor(unsigned char *state){
	for(int i=0;i < 16;i++)
	{
	   state[i] = state[i]^key[i];
	}
}



#pragma pack(push, 1)
typedef struct tagBITMAPFILEHEADER
{
    WORD bfType;  //specifies the file type
    DWORD bfSize;  //specifies the size in bytes of the bitmap file
    WORD bfReserved1;  //reserved; must be 0
    WORD bfReserved2;  //reserved; must be 0
    DWORD bOffBits;  //species the offset in bytes from the bitmapfileheader to the bitmap bits
}BITMAPFILEHEADER;
#pragma pack(pop)


#pragma pack(push, 1)
typedef struct tagBITMAPINFOHEADER
{
    DWORD biSize;  //specifies the number of bytes required by the struct
    LONG biWidth;  //specifies width in pixels
    LONG biHeight;  //species height in pixels
    WORD biPlanes; //specifies the number of color planes, must be 1
    WORD biBitCount; //specifies the number of bit per pixel
    DWORD biCompression;//spcifies the type of compression
    DWORD biSizeImage;  //size of image in bytes
    LONG biXPelsPerMeter;  //number of pixels per meter in x axis
    LONG biYPelsPerMeter;  //number of pixels per meter in y axis
    DWORD biClrUsed;  //number of colors used by th ebitmap
    DWORD biClrImportant;  //number of colors that are important
}BITMAPINFOHEADER;
#pragma pack(pop)


__global__ void RB_Swap(unsigned char *imageData, int size)
{
    int imageIdx = threadIdx.x+blockIdx.x*blockDim.x;

    if(imageIdx<size/3)
    {
        unsigned char tempRGB;
        imageIdx = imageIdx*3;
        tempRGB = imageData[imageIdx];
        imageData[imageIdx] = imageData[imageIdx + 2];
        imageData[imageIdx + 2] = tempRGB;
    }
}



unsigned char *LoadBitmapFile(char *filename, BITMAPINFOHEADER *bitmapInfoHeader, BITMAPFILEHEADER *bitmapFileHeader)
{
    FILE *filePtr; //our file pointer
    unsigned char *bitmapImage;  //store image data

    //open filename in read binary mode
    filePtr = fopen(filename,"rb");
    if (filePtr == NULL)
        return NULL;

    //read the bitmap file header
    fread(bitmapFileHeader, sizeof(BITMAPFILEHEADER),1,filePtr);

    
    //verify that this is a bmp file by check bitmap id
    if (bitmapFileHeader->bfType !=0x4D42)
    {
        fclose(filePtr);
        return NULL;
    }
    
    //read the bitmap info header
    fread(bitmapInfoHeader, sizeof(BITMAPINFOHEADER),1,filePtr); // small edit. forgot to add the closing bracket at sizeof

    //move file point to the begging of bitmap data
    fseek(filePtr, bitmapFileHeader->bOffBits, SEEK_SET);

    //allocate enough memory for the bitmap image data
    bitmapImage = (unsigned char*)malloc(bitmapInfoHeader->biSizeImage);

    //verify memory allocation
    if (!bitmapImage)
    {
        free(bitmapImage);
        fclose(filePtr);
        return NULL;
    }

    //read in the bitmap image data
    fread(bitmapImage,1,bitmapInfoHeader->biSizeImage,filePtr);

    //make sure bitmap image data was read
    if (bitmapImage == NULL)
    {
        fclose(filePtr);
        return NULL;
    }

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t end;
    hipEventCreate(&end);
    float swapTime;

    //swap the r and b values to get RGB (bitmap is BGR)    
    unsigned char *d_bitmapImage;  //store image data in device
    
    //Allocate size to array in device memory
    hipMalloc((void**)&d_bitmapImage, bitmapInfoHeader->biSizeImage);

    //Copy data from host to device
    hipMemcpy(d_bitmapImage, bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyHostToDevice);

    int B = ceil(bitmapInfoHeader->biSizeImage/(1024*16));
    int T = 1024;

    //Kernel call
    hipEventRecord(start, 0);
    //RB_Swap<<<B, T>>> (d_bitmapImage, bitmapInfoHeader->biSizeImage);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipMemcpy(bitmapImage, d_bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&swapTime, start, end);
    printf("Load Swap Time: %fms\n",swapTime);

    hipEventDestroy(start);
    hipEventDestroy(end);

    //close file and return bitmap iamge data
    fclose(filePtr);
    return bitmapImage;
}

void ReloadBitmapFile(char *filename, unsigned char *bitmapImage, BITMAPFILEHEADER *bitmapFileHeader, BITMAPINFOHEADER *bitmapInfoHeader)
{
    FILE *filePtr; //our file pointer

    //open filename in write binary mode
    filePtr = fopen(filename,"wb");
    if (filePtr == NULL)
    {
        printf("\nERROR: Cannot open file %s", filename);
        exit(1);
    }
        

    //write the bitmap file header
    fwrite(bitmapFileHeader, sizeof(BITMAPFILEHEADER),1,filePtr);

    //write the bitmap info header
    fwrite(bitmapInfoHeader, sizeof(BITMAPINFOHEADER),1,filePtr); // small edit. forgot to add the closing bracket at sizeof

    //swap the r and b values to get RGB (bitmap is BGR)

    unsigned char *d_bitmapImage;  //store image data in device
    
    //Allocate size to array in device memory
    hipMalloc((void**)&d_bitmapImage, bitmapInfoHeader->biSizeImage);

    //Copy data from host to device
    hipMemcpy(d_bitmapImage, bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyHostToDevice);

    int B = ceil(bitmapInfoHeader->biSizeImage/(1024*16));
    int T = 1024;

    //Kernel call
    
    //RB_Swap<<<B, T>>> (d_bitmapImage, bitmapInfoHeader->biSizeImage);

    hipMemcpy(bitmapImage, d_bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyDeviceToHost);

    //write in the bitmap image data
    fwrite(bitmapImage,bitmapInfoHeader->biSizeImage,1,filePtr);

    //close file
    fclose(filePtr);
}

__global__ void encrypt(unsigned char *bitmapImage, int size, int threadN)
{
    int threadId = threadIdx.x + blockIdx.x*blockDim.x;
    __shared__ unsigned char sdata[512*16];
    int i;
    unsigned int tid = threadIdx.x;

    for(int k = tid * 16; k < (tid + 1) * 16; k++){
	int gid = k + blockIdx.x * 512 * 16;
        if(gid < size)
	   sdata[k] = bitmapImage[gid];
    }
    __syncthreads();

   
    //substitution

    //unsigned char *p = bitmapImage;
    for(i = tid * 16; i < (tid+1) * 16; i++){
	sdata[i] = s_box[sdata[i]];
    }
    __syncthreads();

    //shift rows
    shift_rows(&sdata[tid * 16]);
    __syncthreads();

    //mix columns
    mix_columns(&sdata[tid * 16]);
    __syncthreads();

    //key_xor
    key_xor(&sdata[tid * 16]);
    __syncthreads();

    for(int k = tid * 16; k < (tid + 1) * 16; k++){
	int gid = k + blockIdx.x * 512 * 16;
        if(gid < size)
	   bitmapImage[gid] = sdata[k];
    }
    __syncthreads();

}

__global__ void decrypt(unsigned char *bitmapImage, int size, int threadN)
{
    int threadId = threadIdx.x + blockIdx.x*blockDim.x;
    __shared__ unsigned char sdata[512*16];
    int i;
    //unsigned char *p = bitmapImage;
    unsigned int tid = threadIdx.x;

    for(int k = tid * 16; k < (tid + 1) * 16; k++){
	int gid = k + blockIdx.x * 512 * 16;
        if(gid < size)
	   sdata[k] = bitmapImage[gid];
    }
    __syncthreads();

    //key_xor
    key_xor(&sdata[tid * 16]);
    __syncthreads();

    //mix columns
    inv_mix_columns(&sdata[tid * 16]);
    __syncthreads();

    //substitution
    for(i = tid * 16; i < (tid+1) * 16; i++){
	sdata[i] = inv_s_box[sdata[i]];
    }
    __syncthreads();

    if(threadId * 16 < size)
	inv_shift_rows(&sdata[tid * 16]);
    __syncthreads();

    for(int k = tid * 16; k < (tid + 1) * 16; k++){
	int gid = k + blockIdx.x * 512 * 16;
        if(gid < size)
	   bitmapImage[gid] = sdata[k];
    }
    __syncthreads();

}

int main()
{
    BITMAPINFOHEADER bitmapInfoHeader;
    BITMAPFILEHEADER bitmapFileHeader;
    unsigned char *bitmapData;
    bitmapData = LoadBitmapFile("lena.bmp",&bitmapInfoHeader, &bitmapFileHeader);
    printf("%d\n",bitmapInfoHeader.biSizeImage);
    
    /*
    //Print array to file
    FILE *fout = fopen("out.bmp","wb");
    fwrite(bitmapData,bitmapInfoHeader.biSizeImage,1,fout);
    */

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t end;
    hipEventCreate(&end);
    float encryptionTime, decryptionTime, HostToDevice, DeviceToHost;
    
    //Encryption

    int key = 8000;

    unsigned char *d_bitmapImage;  //store image data in device
    
    //Allocate size to array in device memory
    hipMalloc((void**)&d_bitmapImage, bitmapInfoHeader.biSizeImage);

    //Copy data from host to device
    hipEventRecord(start, 0);
    hipMemcpy(d_bitmapImage, bitmapData, bitmapInfoHeader.biSizeImage, hipMemcpyHostToDevice);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&HostToDevice, start, end);
    printf("Host to Device Time: %fms\n",HostToDevice);

    int B = ceil(bitmapInfoHeader.biSizeImage/(512*16));
    int T = 512;
    int threadN = B * T;

    //Kernel call
    hipEventRecord(start, 0);
    encrypt<<<B, T>>> (d_bitmapImage, bitmapInfoHeader.biSizeImage, threadN);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&encryptionTime, start, end);
    printf("Encryption Time: %fms\n",encryptionTime);

    //Copy data from device to host
    hipEventRecord(start, 0);
    hipMemcpy(bitmapData, d_bitmapImage, bitmapInfoHeader.biSizeImage, hipMemcpyDeviceToHost);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&DeviceToHost, start, end);
    printf("Device to Host Time: %fms\n",DeviceToHost);
    
    ReloadBitmapFile("Encrypted.bmp", bitmapData, &bitmapFileHeader, &bitmapInfoHeader);

    //load encrypted image to array
    bitmapData = LoadBitmapFile("Encrypted.bmp",&bitmapInfoHeader, &bitmapFileHeader);

    //Decryption
    hipMemcpy(d_bitmapImage, bitmapData, bitmapInfoHeader.biSizeImage, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    decrypt<<<B, T>>> (d_bitmapImage, bitmapInfoHeader.biSizeImage, threadN);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&decryptionTime, start, end);
    printf("Decryption Time: %fms\n",decryptionTime);
    hipMemcpy(bitmapData, d_bitmapImage, bitmapInfoHeader.biSizeImage, hipMemcpyDeviceToHost);

    //decrypt(bitmapData, bitmapInfoHeader.biSizeImage);

    ReloadBitmapFile("Decrypted.bmp", bitmapData, &bitmapFileHeader, &bitmapInfoHeader);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_bitmapImage);

    return 0;
}
